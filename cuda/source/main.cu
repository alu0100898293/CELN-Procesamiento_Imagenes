#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cmath>
#include <cassert>
#include <SFML/Graphics/Image.hpp>
#include <hip/hip_runtime.h>
#include <>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

__device__ void alignChannel(int& channelValue)
{
    channelValue = (channelValue > 255) ? 255 : channelValue;
    channelValue = (channelValue < 0) ? 0 : channelValue;
}

__global__ void applyFilterOnCuda(
    const sf::Uint8* inputImageData, sf::Uint8* outputImageData,
    const std::size_t width, const std::size_t height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    const int Gx[3][3] = {
        {-1, 0, 1},
        {-2, 0, 2},
        {-1, 0, 1} };

    const int Gy[3][3] = {
        {-1, -2, -1},
        {0,  0, 0  },
        {1,  2, 1  } };

    if (((x - 1) > 0 && (x + 1) < width) &&
        ((y - 1) > 0 && (y + 1) < height))
    {
        int newRedChannel{}, newGreenChannel{}, newBlueChannel{};
        int GxRed{}, GyRed{}, GxBlue{}, GyBlue{}, GxGreen{}, GyGreen{};
        for (int kernelX = -1; kernelX <= 1; ++kernelX)
        {
            for (int kernelY = -1; kernelY <= 1; ++kernelY)
            {
                const auto GxValue = Gx[kernelX + 1][kernelY + 1];
                const auto GyValue = Gy[kernelX + 1][kernelY + 1];

                const auto pixel = &inputImageData[((x + kernelX) + (y + kernelY) * width) * 4];

                //Gradient X
                GxRed += static_cast<int>(pixel[0] * GxValue);
                GxGreen += static_cast<int>(pixel[1] * GxValue);
                GxBlue += static_cast<int>(pixel[2] * GxValue);

                //Gradient Y
                GyRed += static_cast<int>(pixel[0] * GyValue);
                GyGreen += static_cast<int>(pixel[1]* GyValue);
                GyBlue += static_cast<int>(pixel[2] * GyValue);
            }
        }

        newRedChannel = abs(GxRed) + abs(GyRed);
        newGreenChannel = abs(GxGreen) + abs(GyGreen);
        newBlueChannel = abs(GxBlue) + abs(GyBlue);
        
        alignChannel(newRedChannel);
        alignChannel(newGreenChannel);
        alignChannel(newBlueChannel);

        auto outPixel = &outputImageData[(x + y * width) * 4];
        outPixel[0] = newRedChannel;
        outPixel[1] = newGreenChannel;
        outPixel[2] = newBlueChannel;
    }
}

auto calculateImageSize(const sf::Image& image)
{
    return image.getSize().x * image.getSize().y * 4;
}

void applyFilter(sf::Image& image, int block)
{
    thrust::host_vector<sf::Uint8> hostImageData{ image.getPixelsPtr(), image.getPixelsPtr() + calculateImageSize(image) };
    thrust::device_vector<sf::Uint8> devImageData(calculateImageSize(image));
    thrust::device_vector<sf::Uint8> devOutputImageData(calculateImageSize(image));
    thrust::copy(hostImageData.begin(), hostImageData.end(), devImageData.begin());

    dim3 dimBlock(block, block);
    dim3 dimGrid(static_cast<uint32_t>(ceil((float)image.getSize().x / dimBlock.x)),
                   static_cast<uint32_t>(ceil((float)image.getSize().y / dimBlock.y)));

    applyFilterOnCuda<<<dimGrid, dimBlock>>>(
        devImageData.data().get(), devOutputImageData.data().get(),
        image.getSize().x, image.getSize().y);

    thrust::copy(devOutputImageData.begin(), devOutputImageData.end(), hostImageData.begin());
    image.create(image.getSize().x, image.getSize().y, hostImageData.data());
}


sf::Image  loadImage(std::string &imageName)
{
    sf::Image image{};
    
    if(imageName.empty()){
        std::cout << "No image specified, using default" << std::endl;
        imageName = "hoja.jpg";
    }
        
    image.loadFromFile("../images/" + imageName);
    
    return image;
}

void saveImage(sf::Image& image,std::string imageName)
{
    image.saveToFile("./out/" + imageName);
}

int main(int argc, char** argv)
{
    // Timers
    hipEvent_t start, stop;
    
    std::string imageName;
    int block;
    /* Command line parameters processing */
    switch(argc) {
        case 3: 
                imageName = argv[1];
                block = atoi(argv[2]);
                break;
        default: 
                printf("\nUse: %s <Img_Name>  <Dim_Block>", argv[0]);
                break;
    }
    

    sf::Image image = loadImage(imageName);

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    applyFilter(image, block);

    hipEventRecord(stop);
    hipDeviceSynchronize();
    hipEventSynchronize(stop);

    float timeMs{};
    hipEventElapsedTime(&timeMs, start, stop);
    std::cout << "Time: " << timeMs << " ms" << std::endl;

    saveImage(image, imageName);


    return EXIT_SUCCESS;
}